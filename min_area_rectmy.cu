#include "hip/hip_runtime.h"
#include"base.h"

#define BLOCK                      32
#define PI                         3.1415926535897932384626433832795
//tl and bt points coords + area + angle
#define _MIN_AREA_EACH_ANGLE_STRID 6
// max rotate degree of contour points
#define _MAX_ROTATE_DEGREES        90

__global__ void calculateRotateCoef(float2* aCoeffs, const int degrees)
{
    int angle = blockIdx.x * blockDim.x + threadIdx.x;
    if (angle < degrees)
    {
        aCoeffs[angle].x = cos((float)angle *PI );
        aCoeffs[angle].y = sin((float)angle *PI );
    }
}

//
__global__ void resetRotatedPointsBuf(int* rotatedPointsTensor, const int numOfDegrees)
{
    // int pointIdx    = blockIdx.x * blockDim.x + threadIdx.x;
    int contourIdx = blockIdx.x;
    int angleIdx = threadIdx.x;
    if (angleIdx < numOfDegrees)
    {
        auto rotatedPointsTensorp = rotatedPointsTensor + contourIdx * _MAX_ROTATE_DEGREES * _MIN_AREA_EACH_ANGLE_STRID + angleIdx * _MIN_AREA_EACH_ANGLE_STRID;

        rotatedPointsTensorp[0] = INT_MAX;
        rotatedPointsTensorp[1] = INT_MAX;
        rotatedPointsTensorp[2] = INT_MIN;
        rotatedPointsTensorp[3] = INT_MIN;
        rotatedPointsTensorp[4] = -1;
        rotatedPointsTensorp[5] = -1;
    }
}

__global__ void calculateRotateArea(int2* inContourPointsData,
    int* rotatedPointsTensor, float2* rotateCoeffs,
    int* numPointsInContourBuf, int maxNumPointsInContour)
{
    int pointIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int contourIdx = blockIdx.y;

    int angleIdx = blockIdx.z;
    extern __shared__ float rotateCoeffs_sm[];
    rotateCoeffs_sm[2 * angleIdx] = rotateCoeffs[angleIdx].x;
    rotateCoeffs_sm[2 * angleIdx + 1] = rotateCoeffs[angleIdx].y;

    __syncthreads();

    if (pointIdx < numPointsInContourBuf[contourIdx])
    {
        auto rotatedPointsTensorp = rotatedPointsTensor + contourIdx * _MAX_ROTATE_DEGREES * _MIN_AREA_EACH_ANGLE_STRID + angleIdx * _MIN_AREA_EACH_ANGLE_STRID;

        auto px = inContourPointsData[contourIdx * maxNumPointsInContour + pointIdx].x;
        auto py = inContourPointsData[contourIdx * maxNumPointsInContour + pointIdx].y;
        float cos_coeff = rotateCoeffs_sm[2 * angleIdx];
        float sin_coeff = rotateCoeffs_sm[2 * angleIdx + 1];
        int   px_rot = (px * cos_coeff) - (py * sin_coeff);
        int   py_rot = (px * sin_coeff) + (py * cos_coeff);
        //xmin
        atomicMin(&rotatedPointsTensorp[0], px_rot);
        //ymin
        atomicMin(&rotatedPointsTensorp[1], py_rot);
        //xmax
        atomicMax(&rotatedPointsTensorp[2], px_rot);
        //ymax
        atomicMax(&rotatedPointsTensorp[3], py_rot);

        __threadfence();
        int rectWidth
            = rotatedPointsTensorp[2] - rotatedPointsTensorp[0];
        int rectHeight
            = rotatedPointsTensorp[3] - rotatedPointsTensorp[1];
        rotatedPointsTensorp[4] = rectWidth * rectHeight;
        rotatedPointsTensorp[5] = angleIdx;
    }
}


__global__ void findMinAreaAndAngle(int* rotatedPointsTensor, float* outMinAreaRectBox,
    const int numOfDegrees)
{
    int angleIdx = threadIdx.x;
    if (angleIdx > numOfDegrees)
    {
        return;
    }

    int rectIdx = blockIdx.x;
    extern __shared__ int areaAngleBuf_sm[];
    auto rotatedPointsTensorp = rotatedPointsTensor + rectIdx * _MAX_ROTATE_DEGREES * _MIN_AREA_EACH_ANGLE_STRID + angleIdx * _MIN_AREA_EACH_ANGLE_STRID;
    areaAngleBuf_sm[2 * angleIdx] = rotatedPointsTensorp[4];
    areaAngleBuf_sm[2 * angleIdx + 1] = rotatedPointsTensorp[5];
    __syncthreads();

    for (int stride = numOfDegrees / 2; stride > 0; stride >>= 1)
    {
        if (angleIdx < stride)
        {
            int* curAreaIdx = &areaAngleBuf_sm[2 * angleIdx];
            int* nextAreaIdx = &areaAngleBuf_sm[2 * (angleIdx + stride)];
            int* curAngleIdx = &areaAngleBuf_sm[2 * angleIdx + 1];
            int* nextAngleIdx = &areaAngleBuf_sm[2 * (angleIdx + stride) + 1];
            if (*curAreaIdx > *nextAreaIdx)
            {
                *curAreaIdx = *nextAreaIdx;
                *curAngleIdx = *nextAngleIdx;
            }
        }
        __syncthreads();

        if (stride % 2 == 1 && areaAngleBuf_sm[0] > areaAngleBuf_sm[2 * (stride - 1)])
        {
            areaAngleBuf_sm[0] = areaAngleBuf_sm[2 * (stride - 1)];
            areaAngleBuf_sm[1] = areaAngleBuf_sm[2 * (stride - 1) + 1];
        }
        __syncthreads();
    }
    if (numOfDegrees % 2 == 1 && areaAngleBuf_sm[0] > areaAngleBuf_sm[2 * (numOfDegrees - 1)])
    {
        areaAngleBuf_sm[0] = areaAngleBuf_sm[2 * (numOfDegrees - 1)];
        areaAngleBuf_sm[1] = areaAngleBuf_sm[2 * (numOfDegrees - 1) + 1];
    }
    if (threadIdx.x == 0)
    {
        int   minRotateAngle = areaAngleBuf_sm[1];
        float cos_coeff = cos(-minRotateAngle * PI / 180.f);
        float sin_coeff = sin(-minRotateAngle * PI / 180.f);
        
        auto rotatedPointsTensorT= rotatedPointsTensor + rectIdx * _MAX_ROTATE_DEGREES * _MIN_AREA_EACH_ANGLE_STRID + areaAngleBuf_sm[1] * _MIN_AREA_EACH_ANGLE_STRID;
        
        float xmin = rotatedPointsTensorT[0];
        float ymin = rotatedPointsTensorT[1];
        float xmax = rotatedPointsTensorT[2];
        float ymax = rotatedPointsTensorT[3];

        float tl_x = (xmin * cos_coeff) - (ymin * sin_coeff);
        float tl_y = (xmin * sin_coeff) + (ymin * cos_coeff);
        float br_x = (xmax * cos_coeff) - (ymax * sin_coeff);
        float br_y = (xmax * sin_coeff) + (ymax * cos_coeff);
        float tr_x = (xmax * cos_coeff) - (ymin * sin_coeff);
        float tr_y = (xmax * sin_coeff) + (ymin * cos_coeff);
        float bl_x = (xmin * cos_coeff) - (ymax * sin_coeff);
        float bl_y = (xmin * sin_coeff) + (ymax * cos_coeff);

        auto outMinAreaRectBoxP = outMinAreaRectBox + rectIdx * 8;

        outMinAreaRectBoxP[0] = bl_x;
        outMinAreaRectBoxP[1] = bl_y;
        outMinAreaRectBoxP[2] = tl_x;
        outMinAreaRectBoxP[3] = tl_y;
        outMinAreaRectBoxP[4] = tr_x;
        outMinAreaRectBoxP[5] = tr_y;
        outMinAreaRectBoxP[6] = br_x;
        outMinAreaRectBoxP[7] = br_y;
    }
}


void calculateRotateCoefCUDA(float2* rotateCoefBuf, const int degrees, const hipStream_t& stream)
{
    dim3 block(BLOCK * 8);
    dim3 grid(cv::divUp(degrees, block.x));
    calculateRotateCoef << <grid, block, 0, stream >> > (rotateCoefBuf, degrees);
}


void MinAreaRectF(std::vector<std::vector<cv::Point>>contours, std::vector<std::vector<float>>& result, hipStream_t stream)
{

    // rotateCoeffsData ÿ��angle �� cos ֵ�� sin ֵ
    float2* rotateCoeffsData = 0;
    hipMalloc((void**)&rotateCoeffsData, sizeof(float) * _MAX_ROTATE_DEGREES * 2);
    calculateRotateCoefCUDA(rotateCoeffsData, _MAX_ROTATE_DEGREES, stream);

    // contours�����㼯��
    int maxNumPointsInContour = 0;

    for (int id = 0; id < contours.size(); id++)
    {
        maxNumPointsInContour = std::max(maxNumPointsInContour, (int)contours[id].size());
    }

    // contour����
    int contourBatch = contours.size();

    // pointsInContourData ����contour��ÿ��ĵ㼯��
    int* numPointsInContourH = (int*)malloc(sizeof(int) * contourBatch);
    // contour�ڵĵ㼯
    int2* inContourPointsDataH = (int2*)malloc(sizeof(int) * maxNumPointsInContour * contourBatch * 2);
    memset(inContourPointsDataH,0,sizeof(int) * maxNumPointsInContour * contourBatch * 2);
    for (int id = 0; id < contours.size(); id++)
    {
        auto contour = contours[id];
        auto inContourPointsDataHP = inContourPointsDataH + id * maxNumPointsInContour;
        for (int pid = 0; pid < contour.size(); pid++)
        {

            inContourPointsDataHP[pid].x = contour[pid].x;
            inContourPointsDataHP[pid].y = contour[pid].y;
        }
        numPointsInContourH[id] = (int)contour.size();
    }
    // pointsInContourData ����contour��ÿ��ĵ㼯��
    int* pointsInContourData = 0;
    hipMalloc((void**)&pointsInContourData, sizeof(int) * contourBatch);
    hipMemcpy(pointsInContourData, numPointsInContourH, sizeof(int) * contourBatch, hipMemcpyHostToDevice);
    // contour�ڵĵ㼯
    int2* inContourPointsData = 0;
    hipMalloc((void**)&inContourPointsData, sizeof(int) * maxNumPointsInContour * contourBatch * 2);
    hipMemcpy(inContourPointsData, inContourPointsDataH, sizeof(int) * maxNumPointsInContour * contourBatch * 2, hipMemcpyHostToDevice);

    // ������
    float* outMinAreaRectData = 0;
    hipMalloc((void**)&outMinAreaRectData, sizeof(float) * contourBatch * 8);

    int* rotatedPointsTensor = 0;
    hipMalloc((void**)&rotatedPointsTensor, sizeof(int) * contourBatch * _MAX_ROTATE_DEGREES * _MIN_AREA_EACH_ANGLE_STRID);

    dim3 block1(128);
    dim3 grid1(contourBatch);
    resetRotatedPointsBuf << <grid1, block1, 0, stream >> > (rotatedPointsTensor, _MAX_ROTATE_DEGREES);

    dim3   block2(256);
    dim3   grid2(cv::divUp(maxNumPointsInContour, block2.x), contourBatch, _MAX_ROTATE_DEGREES);
    size_t smem_size = 2 * _MAX_ROTATE_DEGREES * sizeof(float);
    calculateRotateArea << <grid2, block2, smem_size, stream >> > (inContourPointsData, rotatedPointsTensor,
        rotateCoeffsData, pointsInContourData, maxNumPointsInContour);

    hipStreamSynchronize(stream);

    dim3 grid3(contourBatch);
    findMinAreaAndAngle << <grid3, block2, smem_size, stream >> > (rotatedPointsTensor, outMinAreaRectData,
        _MAX_ROTATE_DEGREES);

    hipStreamSynchronize(stream);

    float* resultList = (float*)malloc(sizeof(float) * 8 * contourBatch);
    hipMemcpy(resultList, outMinAreaRectData, sizeof(float) * 8 * contourBatch, hipMemcpyDeviceToHost);

    for (int cid = 0; cid < contourBatch; cid++)
    {
        auto resultListP = resultList + cid * 8;
        for (int i = 0; i < 8; i++)
        {
            result[cid][i] = resultListP[i];
            printf(" %.2f ", resultListP[i]);
        }
        printf("\n");
    }

    free(numPointsInContourH);
    free(inContourPointsDataH);
    hipFree(inContourPointsData);
    hipFree(rotatedPointsTensor);
    hipFree(outMinAreaRectData);
    hipFree(rotateCoeffsData);

}





